#include "hip/hip_runtime.h"
#include "core.h"

__global__ void kernel(int size, float *input, float *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = logf(1 / (1 + input[idx]));
    }
}

void cudaCallback(int width, int height, float *sample, float **result) {
    int size = width * height;
    float *input_d, *output_d;

    // Allocate device memory and copy data from host to device
    CHECK(hipMalloc((void **)&input_d, sizeof(float)*size));
    CHECK(hipMalloc((void **)&output_d, sizeof(float)*size));
    CHECK(hipMemcpy(input_d, sample, sizeof(float)*size, hipMemcpyHostToDevice));

    // Invoke the device function
    kernel<<< divup(size, 1024), 1024 >>>(size, input_d, output_d);
    hipDeviceSynchronize();

    // Copy back the results and de-allocate the device memory
    *result = (float *)malloc(sizeof(float)*size);
    CHECK(hipMemcpy(*result, output_d, sizeof(float)*size, hipMemcpyDeviceToHost));
    CHECK(hipFree(input_d));
    CHECK(hipFree(output_d));

    // Note that you don't have to free sample and *result by yourself
}